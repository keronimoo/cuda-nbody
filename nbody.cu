#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

typedef struct {
    float x, y, z, vx, vy, vz;
} Particle;

void randomizeParticles(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__
void calculateForces(Particle *particles, float dt, int n) {
    int start = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = start; i < n; i += stride) {
        float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = particles[j].x - particles[i].x;
            float dy = particles[j].y - particles[i].y;
            float dz = particles[j].z - particles[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        particles[i].vx += dt * Fx;
        particles[i].vy += dt * Fy;
        particles[i].vz += dt * Fz;
    }
}

__global__
void integratePositions(Particle *particles, float dt, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        particles[i].x += particles[i].vx * dt;
        particles[i].y += particles[i].vy * dt;
        particles[i].z += particles[i].vz * dt;
    }
}

int main(const int argc, const char **argv) {
    int numParticles = 2 << 11;
    if (argc > 1) numParticles = 2 << atoi(argv[1]);

    const char *initializedFile;
    const char *solutionFile;

    if (numParticles == 2 << 11) {
        initializedFile = "09-nbody/files/initialized_4096";
        solutionFile = "09-nbody/files/solution_4096";
    } else {
        initializedFile = "09-nbody/files/initialized_65536";
        solutionFile = "09-nbody/files/solution_65536";
    }

    if (argc > 2) initializedFile = argv[2];
    if (argc > 3) solutionFile = argv[3];

    const float timeStep = 0.01f;
    const int numIterations = 10;

    int particleBytes = numParticles * sizeof(Particle);
    float *particleData;

    hipMallocManaged(&particleData, particleBytes);

    Particle *particles = (Particle *)particleData;

    read_values_from_file(initializedFile, particleData, particleBytes);

    double totalTime = 0.0;

    int deviceId;
    int numSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    hipMemPrefetchAsync(particleData, particleBytes, deviceId);

    for (int iter = 0; iter < numIterations; iter++) {
        StartTimer();

        int numThreads = 128;
        int numBlocks = 32 * numSMs;

        calculateForces<<<numBlocks, numThreads>>>(particles, timeStep, numParticles);
        hipDeviceSynchronize();

        integratePositions<<<numBlocks, numThreads>>>(particles, timeStep, numParticles);
        hipDeviceSynchronize();

        const double elapsedSeconds = GetTimer() / 1000.0;
        totalTime += elapsedSeconds;
    }

    double avgTime = totalTime / (double)(numIterations);
    float billionsOfOpsPerSecond = 1e-9 * numParticles * numParticles / avgTime;
    write_values_to_file(solutionFile, particleData, particleBytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);
    hipMemPrefetchAsync(particleData, particleBytes, hipCpuDeviceId);

    hipFree(particleData);

    return 0;
}
